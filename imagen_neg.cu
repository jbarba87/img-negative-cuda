
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <malloc.h>

// Estructura que contiene datos de la imagen
typedef struct image{
	char *data;
	int cols;
	int rows;
	int depth;
} image;



// Funcion ejecutada en la GPU
__global__ void negativo(char *input_image, char *output_image, int nRows, int nCols){

	int r = blockIdx.x + threadIdx.x;
	int i;

	// Cada thread ejecuta una fila completa
	for (i = 0; i < nCols; i++){
		output_image[nCols*r + i] = 255 - input_image[nCols*r + i];
	}

}

// Functiones utiles
int read_pgm(char *input_name, image *img);
int write_pgm(char *output_name, image *img);



// Function main
int main(int argc, char **argv){

	image lena;

	// Lee imagen
	read_pgm(argv[1], &lena);

	char *pt = lena.data;
	int ndata = lena.rows*lena.cols;

/*
	//	Programa en C
	int i;
	for (i = 0; i < lena.rows*lena.cols; i++){
		pt[i] = 255 - pt[i];
	}
*/

	// Programa en CUDA

	char *device_input_image = NULL;
	char *device_output_image = NULL;
	hipMalloc((void **) &device_input_image , ndata*sizeof(char));
	hipMalloc((void **) &device_output_image , ndata*sizeof(char));

	hipMemcpy(device_input_image, pt, ndata*sizeof(char), hipMemcpyHostToDevice);

	// Invocando a la funcion (blocks, threads)
	int nBlocks = lena.rows/512;
	int nThreads = 512;
	negativo<<<nBlocks, nThreads>>>(device_input_image, device_output_image, lena.rows, lena.cols);

	hipMemcpy(pt, device_output_image, ndata*sizeof(char), hipMemcpyDeviceToHost);


	// Guarda imagen
	write_pgm(argv[2], &lena);

	return 0;

}

int read_pgm(char *input_name, image *img){

	FILE *input_fd = fopen(input_name, "r+");

	if(input_fd == NULL) {
		printf("Error al abrir el archivo : %s\n", input_name);
		exit(1);
	}

	char row[256];

	fscanf(input_fd, "%s\n", row);

	if (strncmp(row, "P5", 2) != 0){
		printf("El archivo no es PGM\n");
		exit(1);
	}

	char s_rows[3], s_cols[3], s_depth[3];
	int rows, cols, depth; 

	fscanf(input_fd,"%s\n",row);
	fgets(row, 256, input_fd);
	fscanf(input_fd,"%s\n", s_cols);
	fscanf(input_fd,"%s\n", s_rows);
	fscanf(input_fd,"%s\n", s_depth);


	rows = atoi(s_rows); 	cols = atoi(s_cols);  depth = atoi(s_depth);

	img->data = (char*) malloc(rows*cols);
	img->cols = cols;
	img->rows = rows;
	img->depth = depth;

	fread(img->data, sizeof(char), rows*cols, input_fd);

	fclose(input_fd);

	return 1;
}

int write_pgm(char *output_name, image *img){

	FILE *output_fd;
	int ndata = img->rows*img->cols;

	output_fd = fopen(output_name, "w");

	fprintf(output_fd, "%s\n", "P5");
	fprintf(output_fd, "#\n");
	fprintf(output_fd, "%i %i\n", img->rows, img->cols);
	fprintf(output_fd, "%i\n", img->depth);

	fwrite(img->data, sizeof(char), ndata, output_fd);

	fclose(output_fd);

	return 1;
}




